#include "hip/hip_runtime.h"
#include <iostream>
#include "GLImageCudas.h"
#include "ColorTools_Device.h"
#include "DomaineMaths.h"
#include "CalibreurCudas.h"

__global__ static void mandelBrotAnimation(uchar4* ptrDevPixels, int w, int h, int N, DomaineMaths domainNew, CalibreurCudas calibreur);

__device__ static float mandelbrot(float x, float y, int N);

void launchMandelBrotAnimation(uchar4* ptrDevPixels, int w, int h, int N, const DomaineMaths& domainNew){
    dim3 blockPerGrid = dim3(32, 32, 1);
    dim3 threadPerBlock = dim3(16, 16, 1);

    CalibreurCudas calibreur(0, 0.99f, 0.0f, 0.7f);
    mandelBrotAnimation<<<blockPerGrid,threadPerBlock>>>(ptrDevPixels, w, h, N, domainNew, calibreur);
}

__global__ static void mandelBrotAnimation(uchar4* ptrDevPixels, int w, int h, int N, DomaineMaths domainNew, CalibreurCudas calibreur){
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int nbThreadY = gridDim.y * blockDim.y;
    int nbThreadX = gridDim.x * blockDim.x;
    int nbThreadCuda = nbThreadY * nbThreadX;

    float dx = (float) (domainNew.dx / (float) w);
    float dy = (float) (domainNew.dy / (float) h);

    unsigned char r, g, b;
    int tid = j +  (i * nbThreadX);

    float x, y;

    while(tid < (w * h)){
	int pixelI = tid / w;
	int pixelJ = tid - w * pixelI;

	x = domainNew.x0 + pixelJ * dx;
	y = domainNew.y0 + pixelI * dy;

	float h = mandelbrot(x, y, N);
	if(h == 0){
	    HSB_TO_RVB(0.0, 0.0, 0.0, r, g, b);
	} else {
	    h = calibreur.calibrate(h);
	    HSB_TO_RVB(h, 1.0, 1.0, r, g, b);
	}

	ptrDevPixels[tid].x = r;
	ptrDevPixels[tid].y = g;
	ptrDevPixels[tid].z = b;
	ptrDevPixels[tid].w = 255;

	tid += nbThreadCuda;
    }
}

__device__ static float mandelbrot(float x, float y, int N){
    float imag = 0.0;
    float real = 0.0;

    float n = 0;
    float norm;

    do{
	float tmpReal = real;
	real = real * real - imag * imag + x;
	imag = tmpReal * imag + imag * tmpReal + y;

	++n;

	norm = sqrt(real * real + imag * imag);
    } while (norm <= 2.0 && n < N);

    return n == N ? 0 : (n / (float) N);
}
